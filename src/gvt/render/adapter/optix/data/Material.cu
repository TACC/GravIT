#include "hip/hip_runtime.h"


//#include "cutil_math.h"
//#include "Material.cuh"

#include <gvt/render/data/primitives/Material.h>
#include <gvt/render/adapter/optix/data/CUDAMaterial.h>
#include <gvt/render/adapter/optix/data/Material.cuh>

#include "cutil_math.h"

using namespace gvt::render::data::cuda_primitives;
using namespace gvt::render::data::primitives;

   __device__ float4 lambertShade(const gvt::render::data::primitives::CUDALambert* material,
       const Ray &ray, const float4 &N, const Light *lightSource) {

    float4 hitPoint = ray.origin + ray.direction * ray.t;
    float4 L = normalize(lightSource->light.position - hitPoint);
    float NdotL = fmaxf(0.f, fabs(N * L));
    float4 lightSourceContrib = lightSource->contribution(hitPoint);
    float4 diffuse = prod(lightSourceContrib, material->kd) * (NdotL * ray.w);
    return diffuse;
  }


   __device__ float4 phongShade(const gvt::render::data::primitives::CUDAPhong* material,
		   const Ray &ray, const float4 &N, const Light *lightSource) {


	float4 hitPoint = (float4)ray.origin + (ray.direction * ray.t);
    float4 L = (float4)lightSource->light.position - hitPoint;

    L = normalize(L);
    float NdotL = fmaxf(0.f, (N * L));
    float4 R = ((N * 2.f) * NdotL) - L;
    float VdotR = max(0.f, (R * (-1*ray.direction)));
    float power = VdotR * pow(VdotR, material->alpha);

    float4 lightSourceContrib = lightSource->contribution(hitPoint); //  distance;

    float4 diffuse = prod((lightSourceContrib * NdotL), material->kd) * ray.w;
    float4 specular = prod((lightSourceContrib * power), material->ks) * ray.w;

    float4 finalColor = (diffuse + specular);
    return finalColor;
  }

   __device__ float4 blinnShade(const gvt::render::data::primitives::CUDABlinnPhong* material,
		   const Ray &ray, const float4 &N, const Light *lightSource) {
    float4 hitPoint = (float4)ray.origin + (ray.direction * ray.t);
    float4 L = (float4)lightSource->light.position - hitPoint;
    L = normalize(L);
    float NdotL = fmaxf(0.f, (N * L));

    float4 H = normalize((L - ray.direction));

    float NdotH = (H * N);
    float power = NdotH * pow(NdotH, material->alpha);

    float4 lightSourceContrib = lightSource->contribution(hitPoint);

    float4 diffuse = prod((lightSourceContrib * NdotL), material->kd) * ray.w;
    float4 specular = prod((lightSourceContrib * power), material->ks) * ray.w;

    float4 finalColor = (diffuse + specular);
    return finalColor;
  }


   __device__  float4 gvt::render::data::cuda_primitives::Shade(
          gvt::render::data::primitives::Material* material,
                             const Ray &ray,
                             const float4 &sufaceNormal,
                             const Light *lightSource,
                             const float4 lightPostion)
     {

                 float4 r;
                   switch (material->type) {
                   case CUDA_LAMBERT:
                           r = lambertShade((CUDALambert*)material,
                                            ray, sufaceNormal, lightSource);
                           break;
                   case CUDA_PHONG:
                           r = phongShade((CUDAPhong*)material,
                        		   ray, sufaceNormal, lightSource);
                           break;
                   case CUDA_BLINN:
                           r = blinnShade((CUDABlinnPhong*)material,
                        		   ray, sufaceNormal, lightSource);
                           break;
                   default:
                	   printf("Unknown material type\n");
                           break;
                   }
                 return r;


   };
